#include "hip/hip_runtime.h"
#include "common.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream> // debug

namespace flash {

__global__ void forward_kernel_2d_row_tile(
    float const *__restrict__ Q, // query vector
    float const *__restrict__ K, // key vector
    float const *__restrict__ V, // value vector
    int N,                       // sequence length
    int d,                       // head_dim
    int Bc, int Br,              // column tile size and row tile size
    float softmax_scale,         // 1/sqrt(d)
    float *__restrict__ l,       // storage temp for row \sum exp(S)
    float *__restrict__ m,       // storage temp for row \max S
    float *__restrict__ O)       // output attention
{
  auto batch = blockIdx.x;
  auto head = blockIdx.y;
  auto numHeads = gridDim.y;

  auto qkv_offset = (batch * numHeads * N * d) + (head * N * d);
  auto lm_offset = (batch * numHeads * N) + (head * N);

  extern __shared__ float sram[];
  float *Qi = sram;                    // size = Br x d
  float *Kj = &sram[Br * d];           // size = Bc x d
  float *Vj = &sram[d * (Bc + Br)];    // size = Bc x d
  float *S = &sram[d * (Br + 2 * Bc)]; // size = Br x Bc

  auto const tx = threadIdx.x;
  auto const ty = threadIdx.y;
  auto const warp = ty;
  auto const numWarps = blockDim.y;

  auto const iStart = blockIdx.z * Br;
  auto const iEnd = min(iStart + Br, N);
  auto const Brc = min(Br, N - iStart);

  // set l and m to default values
  for (int i = iStart + ty * blockDim.x + tx; i < iEnd; i += warpSize * numWarps) {
    l[lm_offset + i] = 0.f;
    m[lm_offset + i] = -INFINITY;
  }

  // Load Q tile
  for (int ii = warp; ii < Brc; ii += numWarps) {
    auto i = iStart + ii;
    for (int k = tx; k < d; k += blockDim.x) {
      Qi[ii * d + k] = Q[qkv_offset + i * d + k];
    }
  }

  for (int jStart = 0; jStart < N; jStart += Bc) { // loop j tiles
    // Potentially cropped Bc in the last tile
    auto Bcc = min(Bc, N - jStart);

    // load Kj, Vj
    for (int k = tx; k < d; k += blockDim.x) {
      auto jj = ty;
      auto j = jStart + jj;
      Kj[jj * d + k] = (j < N) ? K[qkv_offset + j * d + k] : 0.f;
      Vj[jj * d + k] = (j < N) ? V[qkv_offset + j * d + k] : 0.f;
    }
    __syncthreads();

    // Compute Sij and row_max
    for (int ii = warp; ii < Brc; ii += numWarps) {
      auto i = iStart + ii;

      float row_m = -INFINITY;
      for (int jj = tx; jj < Bcc; jj += blockDim.x) {
        float Sij = 0.f;
        for (int k = 0; k < d; k++) {
          Sij += Qi[ii * d + k] * Kj[jj * d + k];
        }
        Sij *= softmax_scale;
        S[Bc * ii + jj] = Sij;
        row_m = common::float_max(row_m, Sij);
      }
      row_m = common::warpReduce<common::float_max>(row_m);

      float row_l = 0.f;
      for (int jj = tx; jj < Bcc; jj += blockDim.x) {
        float Sij = __expf(S[Bc * ii + jj] - row_m);
        S[Bc * ii + jj] = Sij;
        row_l += Sij;
      }
      row_l = common::warpReduce<common::float_add>(row_l);

      float row_m_prev = m[lm_offset + i];
      float row_l_prev = l[lm_offset + i];
      float row_m_new = common::float_max(row_m_prev, row_m);
      float row_l_new = __expf(row_m_prev - row_m_new) * row_l_prev +
                        __expf(row_m - row_m_new) * row_l;

      // Product Oik = Pin * Vnk
      // O[Br,d] = S[Br, Bc] * V[Bc, d]
      for (int k = tx; k < d; k += blockDim.x) {
        float PinVnk = 0.f;
        for (int n = 0; n < Bc; n++) {
          PinVnk += S[Bc * ii + n] * Vj[n * d + k];
        }
        O[qkv_offset + i * d + k] =
            ((row_l_prev * __expf(row_m_prev - row_m_new) *
              O[qkv_offset + i * d + k]) +
             (__expf(row_m - row_m_new) * PinVnk)) /
            row_l_new;

        // save new l and m
        if (tx == 0) {
          m[lm_offset + i] = row_m_new;
          l[lm_offset + i] = row_l_new;
        }
      }
    }

    __syncthreads();
  }
}

}
