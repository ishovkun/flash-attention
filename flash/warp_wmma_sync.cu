#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include "cuda_constants.hpp"
#include "common.hpp"

namespace flash {

using namespace nvcuda;

__global__ void warp_wmma_sync(const float *Q, const float *K, const float *V,
                               const int N, const int d, const int Tc,
                               const int Tr, const int Bc, const int Br,
                               const float softmax_scale, float *l, float *m,
                               float *O) {
  int tx = threadIdx.x;
  int batch = blockIdx.x;
  int head = blockIdx.y; // batch and head index

  // Offset into Q,K,V,O,l,m - different for each batch and head
  int qkv_offset = (batch * gridDim.y * N * d) + (head * N * d);     // gridDim.y = nh
  int lm_offset = (batch * gridDim.y * N) + (head * N); // offset for l and m

  // padded dimension d for wmma alignment
  auto dp = common::nextMultiple(d, constants::WMMA_N);

  // Define SRAM for Q,K,V,S
  extern __shared__ float sram[];
  int tile_size = Bc * d; // size of Qi, Kj, Vj
  float *Qi = sram;
  float *Kj = &sram[Br * dp];
  float *Vj = &sram[dp * (Bc + Br)];
  float *Sij = &sram[dp*(Br + 2*Bc)];

  // Initialize l and m
  for (int x = 0; x < N; x += warpSize) {
    if (x + tx < N) {
      l[lm_offset + tx + x] = 0;
      m[lm_offset + tx + x] = -INFINITY;
    }
  }

  for (int j = 0; j < Tc; j++) {

    int const Bcc = min(Bc, N - j * Bc);

    // Load Kj, Vj to SRAM
    for (int jj = 0; jj < Bc; jj++ ) {
      for (auto k = tx; k < dp; k += warpSize) {
        auto inBounds = jj < Bcc && k < d;
        Kj[jj*dp + k] = inBounds ? K[qkv_offset + j*tile_size + jj*d + k] : 0.f;
        Vj[jj*dp + k] = inBounds ? V[qkv_offset + j*tile_size + jj*d + k] : 0.f;
      }
    }
    __syncthreads();

    for (int i = 0; i < Tr; i++) {

      // Load Qi to SRAM
      for (int ii = 0; ii < Br; ii++) {
        for (auto k = tx; k < dp; k += warpSize) {
          auto inBounds = i*Br + ii < N && k < d;
          Qi[ii*dp + k] = inBounds ? Q[qkv_offset + (tile_size * i) + ii*d + k] : 0.f;
        }
      }
      __syncthreads();

      // Load l and m to registers
      float row_m_prev = -INFINITY, row_l_prev = 0.f;
      if (tx < Br) {
        row_m_prev = m[lm_offset + (Br * i) + tx];
        row_l_prev = l[lm_offset + (Br * i) + tx];
      }

      // S = QK^T - tensor cores going brrr
      constants::fragA_t q_frag;    // (16x8) WMMA_M x WMMA_K, row_major
      constants::fragB_cm_t k_frag; // (8x16) WMMA_K x WMMA_N, col_major
      constants::fragC_t s_frag;
      fill_fragment(s_frag, 0.0f);

      for (int k = 0; k < d; k += constants::WMMA_K) {
        wmma::load_matrix_sync(q_frag, Qi + k, dp);
        wmma::load_matrix_sync(k_frag, Kj + k, dp);
        // S_frag += q_frag * k_frag
        wmma::mma_sync(s_frag, q_frag, k_frag, s_frag);
      }
      wmma::store_matrix_sync(Sij, s_frag, constants::WMMA_M,
                              wmma::mem_row_major);

      float row_m = -INFINITY;
      float row_l = 0;
      if (tx < Br && Bc * i + tx < N) {
        // Softmax scaling, row_m = rowmax(S)
        for (int x = 0; x < Bcc; x++) {
          Sij[(Bc * tx) + x] *= softmax_scale;
          row_m = max(row_m, Sij[(Bc * tx) + x]);
        }

        // P = exp(S - row_m), row_l = rowsum(P)
        for (int x = 0; x < Bcc; x++) {
          Sij[(Bc * tx) + x] = __expf(Sij[(Bc * tx) + x] - row_m);
          row_l += Sij[(Bc * tx) + x];
        }
      }

      // PV = Pij * Vj - tensor cores going brrr again
      // using namespace wmma;
      constants::fragA_t p_frag;
      constants::fragB_rm_t v_frag;
      constants::fragC_t pv_frag;

      for (int x = 0; x < d; x += constants::WMMA_M) {
        wmma::fill_fragment(pv_frag, 0.0f);
        for (int k = 0; k < Bc; k += constants::WMMA_K) {
          wmma::load_matrix_sync(p_frag, Sij + k, Bc);
          wmma::load_matrix_sync(v_frag, Vj + x + (k * dp), dp);
          wmma::mma_sync(pv_frag, p_frag, v_frag, pv_frag);
        }
        // store it in unused Qi
        wmma::store_matrix_sync(Qi + x, pv_frag, dp, wmma::mem_row_major);
      }

      if (tx < Br && i * Br + tx < N) {
        // Compute new m and l
        float row_m_new = max(row_m_prev, row_m);
        float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) +
                          (__expf(row_m - row_m_new) * row_l);

        // Write O, l, m to HBM
        for (int x = 0; x < d; x++) {
          O[qkv_offset + (tile_size * i) + (tx * d) + x] =
              (1 / row_l_new) *
              ((row_l_prev * __expf(row_m_prev - row_m_new) *
                O[qkv_offset + (tile_size * i) + (tx * d) + x]) +
               (__expf(row_m - row_m_new) * Qi[(tx * dp) + x]));
        }
        m[lm_offset + (Br * i) + tx] = row_m_new;
        l[lm_offset + (Br * i) + tx] = row_l_new;
      }
    }
    __syncthreads();
  }
}

} // namespace flash
